#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <tiffio.h>
#include <stdint.h>

__global__ void blur(uint8_t *d_out, uint8_t *r_in, uint8_t *g_in, uint8_t *b_in, int width, int height){
	
	int id = (blockIdx.x*blockDim.x)+threadIdx.x;
	
	if(id < width*height){
		int filter_size = 5;
		float r_out = 0, g_out = 0, b_out = 0;
		int count = 0;
		for(int row = id; row < id+(width*filter_size); row+=width){
			for(int col = 0; col < filter_size; col++){
				if(col<(id/width+1)*width && row<width*height){
					r_out += r_in[row+col];
					g_out += g_in[row+col];
					b_out += b_in[row+col];
					count++;
				}
			}
		}
		d_out[id*3] = r_out/count;
		d_out[id*3+1] = g_out/count;
		d_out[id*3+2] = b_out/count;

	}

}

__global__ void transpose(uint8_t *d_in, uint8_t *r, uint8_t *g, uint8_t *b){

	int id = (blockIdx.x*blockDim.x)+threadIdx.x;
	if(id%3 == 0)
		r[id/3] = d_in[id];
	else if(id%3 == 1)
		g[id/3] = d_in[id];
	else
		b[id/3] = d_in[id];

}


int main(int argc, char **argv){
  
	uint32_t    width, height;
	TIFF       *iimage;
	uint16_t    bits_per_sample, photometric;
	uint16_t    planar_config;
	uint16_t    samples_per_pixel;
	int size;

	assert(argc == 3);

	iimage = TIFFOpen(argv[1], "r");
	assert(iimage);
	assert(TIFFGetField(iimage, TIFFTAG_IMAGEWIDTH, &width));
	assert(width > 0);
	assert(TIFFGetField(iimage, TIFFTAG_IMAGELENGTH, &height));
	assert(height > 0);
	assert(TIFFGetField(iimage, TIFFTAG_BITSPERSAMPLE, &bits_per_sample) != 0);
	assert(bits_per_sample == 8);
	assert(TIFFGetField(iimage, TIFFTAG_PHOTOMETRIC, &photometric));
	assert(photometric == PHOTOMETRIC_RGB);
	assert(TIFFGetField(iimage, TIFFTAG_PLANARCONFIG, &planar_config) != 0);
	assert(TIFFGetField(iimage, TIFFTAG_SAMPLESPERPIXEL, &samples_per_pixel));
	assert(samples_per_pixel == 3);

	size = width * height * samples_per_pixel * sizeof(char);

	printf("size is %d\n",size);
	printf("spp is %d\n",samples_per_pixel);
	char     *idata = (char *) malloc(size);
	assert(idata != NULL);

	char     *curr = idata;
	int      count = TIFFNumberOfStrips(iimage);
	size_t in;
	for (int i = 0; i < count; ++i) {
		in = TIFFReadEncodedStrip(iimage, i, curr, -1);
//		assert(in != -1);
//		printf("%li\n", in);
		curr += in;
	}
	TIFFClose(iimage);

	char       *odata = (char *) malloc(size);
	uint8_t* d_in;
	hipMalloc((void**) &d_in, size);
	hipMemcpy(d_in, idata, size, hipMemcpyHostToDevice);
	uint8_t* d_out;
	hipMalloc((void**) &d_out, size);
	uint8_t* r_in;
	uint8_t* g_in;
	uint8_t* b_in;
	hipMalloc((void**) &r_in, width*height);
	hipMalloc((void**) &g_in, width*height);
	hipMalloc((void**) &b_in, width*height);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	transpose<<<size/width, width>>>(d_in, r_in, g_in, b_in);
	blur<<<size/width, width>>>(d_out, r_in, g_in, b_in, width, height);
	hipEventRecord(stop);

	hipMemcpy(odata, d_out, size, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("kernel time is %fms\n", milliseconds);

	assert(odata != NULL);
	TIFF       *oimage = TIFFOpen(argv[2], "w");
	assert(oimage);

	assert(TIFFSetField(oimage, TIFFTAG_IMAGEWIDTH, width));
	assert(TIFFSetField(oimage, TIFFTAG_IMAGELENGTH, height));
	assert(TIFFSetField(oimage, TIFFTAG_BITSPERSAMPLE, bits_per_sample));
	assert(TIFFSetField(oimage, TIFFTAG_COMPRESSION, COMPRESSION_DEFLATE));
	assert(TIFFSetField(oimage, TIFFTAG_PHOTOMETRIC, photometric));
	assert(TIFFSetField(oimage, TIFFTAG_SAMPLESPERPIXEL, samples_per_pixel));
	assert(TIFFSetField(oimage, TIFFTAG_PLANARCONFIG, planar_config));
	assert(TIFFSetField(oimage, TIFFTAG_ROWSPERSTRIP, height));

	size_t    on = size;
	assert(TIFFWriteEncodedStrip(oimage, 0, odata, on) == on);
	TIFFClose(oimage);
	free(idata);
	free(odata);
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
